#include "hip/hip_runtime.h"
 #include "arithmeticTests.h"
#include <stdio.h>

template <typename T>
__global__
void globalMemKernel(int n, int iterateNum, volatile T *x) {
  int thread = blockIdx.x*blockDim.x + threadIdx.x;

  volatile T a = 0;

  for (int i = 0; i < iterateNum; i++) {
    for (int j = 0; j < n; j++) {
      a = x[j];
    }
  }
  x[thread] = a;
}

template <typename T>
__global__
void createData(int n, T *x) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  T a = 1.0;
  if (i < n) {
    x[i] = a;
  }
}

template <typename T>
class MemoryTestBase {
public: 

  T *d_x;
  int n;
  int iterNum;
  int numBlocks;
  int blockSize;
  int numBlockScale;
  int opsPerIteration; //number of operations in one iteration. Not including loop calculations

  MemoryTestBase(int blockSize, int iterNum)
    : iterNum(iterNum), blockSize(blockSize), numBlockScale(360)
  { opsPerIteration = 0;}
  MemoryTestBase(int blockSize, int iterNum, int numBlockScale)
    : iterNum(iterNum), blockSize(blockSize), numBlockScale(numBlockScale)
  { opsPerIteration = 0;}

  ~MemoryTestBase() {
    CUDA_ERROR( hipFree(d_x) );
  }

  void kernelSetup(hipDeviceProp_t deviceProp) {
    numBlocks = deviceProp.multiProcessorCount * numBlockScale;
    n = numBlocks * blockSize;
    CUDA_ERROR( hipMalloc(&d_x, n*sizeof(T)) ); 
    createData<T><<<numBlocks, blockSize>>>(n, d_x);
  }

  //get the number of threads launched in the kernel. Must be 
  //called after kernelSetup() or the neccisary fields may not be initialized
  int getNumThreads() {
    return numBlocks * blockSize;
  }

  //return the number of operations that are executed in the kernel's loop
  //for the specified number of operations.
  //Ex: 6 operations per iteration * 1000000 iterations = 6000000 operations
  int getOpsPerThread() {
    return opsPerIteration * iterNum;
  }

  void runKernel();

  void CUDA_ERROR(hipError_t e) {
    if (e != hipSuccess) {
      printf("cuda error in test class: \"%s\"\n", hipGetErrorString(e));
    }
  } 

};


template <typename T>
class GlobalMemTest : public MemoryTestBase<T> {
public:
  GlobalMemTest(int blockSize, int iterNum) 
      : MemoryTestBase<T>(blockSize, iterNum) 
  {this->opsPerIteration = 6;}
  GlobalMemTest(int blockSize, int iterNum, int numBlockScale) 
      : MemoryTestBase<T>(blockSize, iterNum, numBlockScale) 
  {this->opsPerIteration = 6;}

  void runKernel() {
      globalMemKernel<T><<<this->numBlocks, this->blockSize>>>(this->n, this->iterNum, this->d_x);
  }
};
