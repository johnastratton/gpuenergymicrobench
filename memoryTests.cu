#include "hip/hip_runtime.h"
#include "arithmeticTests.h"
#include <stdio.h>


//------------------ L1 CACHE KERNEL -----------
template <typename T>
__global__
void l1MemKernel(int n, int iterateNum, T *x) {
  return;//TODO
}

//------------------ L2 CACHE KERNEL -----------
template <typename T>
__global__
void l2MemKernel(int n, int iterateNum, T *x) {
  for (int i = 0; i < iterateNum; i++) {
    for (int k = 0; k < n; k++) {
        x[(n-1)-k] = x[k];
    }
  }
}



//------------------ GLOBAL CACHE KERNELS -----------
template <typename T>
__global__
void globalMemKernel(int n, int iterateNum, volatile T *x) {
  int thread = blockIdx.x*blockDim.x + threadIdx.x;

  volatile T a = 0;

  for (int i = 0; i < iterateNum; i++) {
    for (int j = 0; j < n; j++) {
      a = x[j];
    }
  }
  x[thread] = a;
}


//------------------ SHARED MEMORY KERNEL -----------
template <typename T>
__global__
void sharedMemKernel(int n, int iterateNum, T *x) {
  return; //TODO
}



//------------------ INITIALIZE ARRAY FOR KERNEL -----------
template <typename T>
__global__
void createData(int n, T *x) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  // T a = 1.0;
  if (i < n) {
    x[i] = i;
  }
}



template <typename T>
class MemoryTestBase {
public: 

  T *d_x;
  int n;
  int iterNum;
  int numBlocks;
  int blockSize;
  int numBlockScale;
  int opsPerIteration; //number of operations in one iteration. Not including loop calculations

  MemoryTestBase(int blockSize, int iterNum)
    : iterNum(iterNum), blockSize(blockSize), numBlockScale(360)
  { opsPerIteration = 0;}
  MemoryTestBase(int blockSize, int iterNum, int numBlockScale)
    : iterNum(iterNum), blockSize(blockSize), numBlockScale(numBlockScale)
  { opsPerIteration = 0;}

  ~MemoryTestBase() {
    CUDA_ERROR( hipFree(d_x) );
  }

  void kernelSetup(hipDeviceProp_t deviceProp) {
    numBlocks = deviceProp.multiProcessorCount * numBlockScale;
    n = numBlocks * blockSize;
    CUDA_ERROR( hipMalloc(&d_x, n*sizeof(T)) ); 
    createData<T><<<numBlocks, blockSize>>>(n, d_x);
  }

  //get the number of threads launched in the kernel. Must be 
  //called after kernelSetup() or the neccisary fields may not be initialized
  int getNumThreads() {
    return numBlocks * blockSize;
  }

  //return the number of operations that are executed in the kernel's loop
  //for the specified number of operations.
  //Ex: 6 operations per iteration * 1000000 iterations = 6000000 operations
  int getOpsPerThread() {
    return opsPerIteration * iterNum;
  }

  void runKernel();

  void CUDA_ERROR(hipError_t e) {
    if (e != hipSuccess) {
      printf("cuda error in test class: \"%s\"\n", hipGetErrorString(e));
    }
  } 

};


template <typename T>
class L1MemTest : public MemoryTestBase<T> {
public:
  L1MemTest(int blockSize, int iterNum) 
      : MemoryTestBase<T>(blockSize, iterNum) 
  {this->opsPerIteration = 6;}
  L1MemTest(int blockSize, int iterNum, int numBlockScale) 
      : MemoryTestBase<T>(blockSize, iterNum, numBlockScale) 
  {this->opsPerIteration = 6;}

  void runKernel() {
      l1MemKernel<T><<<this->numBlocks, this->blockSize>>>(this->n, this->iterNum, this->d_x);
  }
};


template <typename T>
class L2MemTest : public MemoryTestBase<T> {
public:
  L2MemTest(int blockSize, int iterNum) 
      : MemoryTestBase<T>(blockSize, iterNum) 
  {this->opsPerIteration = 6;}
  L2MemTest(int blockSize, int iterNum, int numBlockScale) 
      : MemoryTestBase<T>(blockSize, iterNum, numBlockScale) 
  {this->opsPerIteration = 6;}

  void runKernel() {
      l2MemKernel<T><<<this->numBlocks, this->blockSize>>>(this->n, this->iterNum, this->d_x);
  }
};

template <typename T>
class GlobalMemTest : public MemoryTestBase<T> {
public:
  GlobalMemTest(int blockSize, int iterNum) 
      : MemoryTestBase<T>(blockSize, iterNum) 
  {this->opsPerIteration = 6;}
  GlobalMemTest(int blockSize, int iterNum, int numBlockScale) 
      : MemoryTestBase<T>(blockSize, iterNum, numBlockScale) 
  {this->opsPerIteration = 6;}

  void runKernel() {
      globalMemKernel<T><<<this->numBlocks, this->blockSize>>>(this->n, this->iterNum, this->d_x);
  }
};

template <typename T>
class SharedMemTest : public MemoryTestBase<T> {
public:
  SharedMemTest(int blockSize, int iterNum) 
      : MemoryTestBase<T>(blockSize, iterNum) 
  {this->opsPerIteration = 6;}
  SharedMemTest(int blockSize, int iterNum, int numBlockScale) 
      : MemoryTestBase<T>(blockSize, iterNum, numBlockScale) 
  {this->opsPerIteration = 6;}

  void runKernel() {
      sharedMemKernel<T><<<this->numBlocks, this->blockSize>>>(this->n, this->iterNum, this->d_x);
  }
};
