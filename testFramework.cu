#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include "addFP32.cu"
#include "baseTestClass.cpp"

//run command: 
  //nvcc testRunner.cu -L/usr/lib64/nvidia -lnvidia-ml -I/usr/local/cuda-7.0/samples/common/inc/ -I/nvmlPower.cpp

template <class K>
class TestRunner {
public:

  //int deviceIDNum: GPU device to do all work/sampling on
  int deviceIDNum = 0;

  //device properties
  hipDeviceProp_t deviceProp;

  //desired filename for output
  const char *outputName; 

  //max number of times to run the test in a row
  int maxTestRuns = 1;

  //hold data from samples in vectors
  std::vector<float> powerData;
  std::vector<int> tempData;


  //the deviceID in nvml format
  nvmlDevice_t nvmlDeviceID;

  //the result code of the previous nvml call
  nvmlReturn_t nvmlResult;

  //CUDA timing events to clalculate test kernel's runtime
  hipEvent_t gpuStart, gpuStop, kernelFinished;

  //TODO: remove later. for development testing
  float *d_x, *d_y;

  //class that holds the kernel to run
  //BaseTestClass testClass;
  K &testClass;

  /*
  constructor
    pass kernel functions and output name?
  */
  //TestRunner(BaseTestClass t, const char *outputName) : outputName(outputName) {
    TestRunner(K &t, const char *outputName) : outputName(outputName) {
    testClass = t;
    nvmlResult = nvmlInit();
    if ( nvmlResult != NVML_SUCCESS )
    {
      printf("NVML Init fail: %s\n", nvmlErrorString(nvmlResult));
      exit(0);
    }

    nvmlResult = nvmlDeviceGetHandleByIndex(deviceIDNum, &nvmlDeviceID);
    if (nvmlResult != NVML_SUCCESS) {
      printf("failed getting device handle by index: %s\n", nvmlErrorString(nvmlResult));
      exit(0);
    }
    hipSetDevice(deviceID);
    hipGetDeviceProperties(&deviceProp, deviceID);
  }

  /*
  start measurement of device's power/temp
    initial setup of nvml and vars
  */
  void setupSampling()
  {

    //reserve about 6 min of sampling space
    powerData.clear();
    powerData.reserve(750000);  
    tempData.clear();
    tempData.reserve(750000);  

    CUDA_ERROR( hipEventCreate(&gpuStart) );
    CUDA_ERROR( hipEventCreate(&gpuStop) );
    CUDA_ERROR( hipEventRecord(gpuStart) );
  }


  /*
  takeMeasurements
    loop that samples until kernel has finished
    return bool - true: good sample graph, false: bad samples
  */
  void runSampling() {
    unsigned int powerLevel, tempLevel;
    CUDA_ERROR( hipEventCreate(&kernelFinished) );
    CUDA_ERROR( hipEventRecord(kernelFinished) ); 

    //TODO: may need to change stream above or just use gpuStart
    while (hipEventQuery(kernelFinished) != hipSuccess) {
    // while (hipEventQuery(kernelFinished) != hipSuccess) {
      powerLevel = getPowerUseage();
      tempLevel = getDeviceTemp();
      powerData.push_back(powerLevel);
      tempData.push_back(tempLevel);
    }
    CUDA_ERROR( hipEventRecord(gpuStop, 0) );
    CUDA_ERROR( hipEventSynchronize(gpuStop) );
  }


  /*
  get a good test reading
    keep running test until:
      good reading && temp was constant
      haven't looped too many times
    warn if test case is not long enough
    print each time test case is run
    if loop runs to maxItr, then print to file anyways
  */
  
  void getGoodSample() {
    unsigned int prevTemp = 0;
    unsigned int curTemp = 0;
    bool badSampleData = true;
    int curRun = 1;

    testClass.kernelSetup(deviceProp);

    while( (badSampleData || std::abs((int)(curTemp - prevTemp)) >= 1) 
                && curRun <= maxTestRuns ) 
    {
      printf("  beginning test run %d\n", curRun);
      
      // int n = 1<<18;
      // CUDA_ERROR( hipMalloc(&d_x, n*sizeof(float)) ); 
      // CUDA_ERROR( hipMalloc(&d_y, n*sizeof(float)) );
      // createData<<<(n+255)/256, 256>>>(n, d_x, d_y);

      setupSampling();
      // runTest();
      testClass.runKernel();
      runSampling();

      badSampleData = !isDataValid();

      prevTemp = curTemp;
      curTemp = getDeviceTemp();
      curRun++;
    }

    if (maxTestRuns < curRun) {
      printf("maxTestRuns exceeded, sample data may be bad. Debug info:\n");
      printf("  curTemp - prevTemp = %d\n", (int)curTemp-prevTemp);
      printf("  badSampleData: %d\n", badSampleData);
    }
  }

  /*
  run test on given kernel
    don't worry about sampling, just test setup/run/cleanup
    called by getGoodSample
  */
  /*void runTest() {
    //TODO: generalize function
    int n = 1<<18; // == exactly blockCount * threads/block
    int iterateNum = 1500000;
    addition6_FP32<<<(n+255)/255, 256>>>(n,iterateNum, d_x, d_y);
  }*/

  /*
  analyze test's sampling
    make sure graph is flat and level
    1%-3% tolerance in graph's height
      maybe allow small downward spikes 5ms in width
      very beg and end can ramp up/down though
  return true(good enough) / false(bad samples)
  */
  bool isDataValid() {
    //TODO
    return false;
  }

  /*
  write data to file
  file is .txt but organized as .csv
    first row: "Power(W), Temperature(ºC), Time(ms), elapsed time(ms), number of samples"
    second row: "xxx, yyy, zzz, aaa, bb"
    third row->end: "xxxx, yyyy, zzzz"
  */
  void dataToFile() {
    float elapsedT;
    CUDA_ERROR( hipEventElapsedTime(&elapsedT, gpuStart, gpuStop));
    float timeBetweenSample = elapsedT / powerData.size();
    
    FILE *fp = fopen(outputName, "w+");
    if (fp == NULL) {
	    printf("Attempt at opening '%s' failed. Error: ", outputName);
	    perror("");
	    printf("Terminating...");
	    exit(0);
  	}
    fprintf(fp, "Power(W), Temperature(ºC), Time(ms), elapsed time(ms), number of samples\n");
    fprintf(fp, "%.3lf, %d, %f, %3.1f, %d\n", powerData[0]/1000.0, tempData[0], timeBetweenSample, elapsedT, (int)powerData.size());
    
    for (int i = 1; i < powerData.size(); i++) {
      fprintf(fp, "%.3lf, %d, %f\n", powerData[i]/1000.0, tempData[0], timeBetweenSample*i );
    }
    fclose(fp);
  }

  /*
  find number of devices
  */
  void numOfDevices() {
    unsigned int deviceCount;
    nvmlResult = nvmlDeviceGetCount(&deviceCount);
    if (NVML_SUCCESS != nvmlResult)
    {
		  printf("Failed to query device count: %s\n", nvmlErrorString(nvmlResult));
		  exit(0);
    }
    printf("There are %d GPUs connected, ensure the correct one is selected\n", deviceCount);
  }

  /*
  get device temp
    return temp as unsigned int
    return 0 if there was a sampling error
  */
  unsigned int getDeviceTemp() {
    unsigned int tempLevel;
    nvmlResult = nvmlDeviceGetTemperature(nvmlDeviceID, NVML_TEMPERATURE_GPU, &tempLevel);
    if (nvmlResult != NVML_SUCCESS) {
      tempLevel = 0;
      printf("NVML temp measurement fail: %s\n", nvmlErrorString(nvmlResult));
    }
    return tempLevel;
  }

  unsigned int getPowerUseage() {
  unsigned int powerLevel;
    nvmlResult = nvmlDeviceGetPowerUsage(nvmlDeviceID, &powerLevel);
    if (nvmlResult != NVML_SUCCESS) {
      printf("NVML power measurement fail: %s\n", nvmlErrorString(nvmlResult));
    }
    return powerLevel;
  }

  void CUDA_ERROR(hipError_t e) {
    if (e != hipSuccess) {
      printf("cuda Error: \"%s\"\n", hipGetErrorString(e));
    }
  } 

}; //end TestRunner


int main() {
  printf("creating AdditionFP32 TestClass\n");
  //BaseTestClass test = AdditionFP32();
  AdditionFP32 test = AdditionFP32();

  printf("creating TestRunner obj\n");
  TestRunner tester = TestRunner<AdditionFP32>(test, "output.txt");
  
  printf("calling getGoodSample\n");
  tester.getGoodSample();

  printf("calling dataToFile\n");
  tester.dataToFile();
  return 0;
}
