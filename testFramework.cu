#include "hip/hip_runtime.h"
#include <vector>
#include <cmath>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <unistd.h>
#include "templateAddFP.cu"

//run command: 
  //nvcc testFramework.cu -L/usr/lib64/nvidia -lnvidia-ml -I/usr/local/cuda-7.0/samples/common/inc/ -I/nvmlPower.cpp

template <class K>
class TestRunner {
public:

  //int deviceIDNum: GPU device to do all work/sampling on
  const int deviceIDNum = 0;

  //max number of times to attempt getting a good sample
  const int maxTestRuns = 15;

  //number of samples to igore from beg and end while analyzing data
  const int ignoreSampleCount = 50;

  //device properties
  hipDeviceProp_t deviceProp;

  //desired filename for output
  const char *outputName; 

  //hold data from samples in vectors
  std::vector<float> powerData;
  std::vector<int> tempData;

  //the deviceID in nvml format
  nvmlDevice_t nvmlDeviceID;

  //the result code of the previous nvml call
  nvmlReturn_t nvmlResult;

  //CUDA timing events to clalculate test kernel's runtime
  hipEvent_t gpuStart, gpuStop, kernelFinished;

  //TODO: remove later. for development testing
  float *d_x, *d_y;

  //class that holds the kernel to run
  K *testClass;

  TestRunner(K *t, const char *outputName) : outputName(outputName) {
    testClass = t;
    
    CUDA_ERROR( hipSetDevice(deviceIDNum) );
    CUDA_ERROR( hipGetDeviceProperties(&deviceProp, deviceIDNum) );
    
    nvmlResult = nvmlInit();
    if ( nvmlResult != NVML_SUCCESS )
    {
      printf("NVML Init fail: %s\n", nvmlErrorString(nvmlResult));
      exit(0);
    }
    nvmlResult = nvmlDeviceGetHandleByIndex(deviceIDNum, &nvmlDeviceID);
    if (nvmlResult != NVML_SUCCESS) {
      printf("failed getting device handle by index: %s\n", nvmlErrorString(nvmlResult));
      exit(0);
    }
  }

  /*
  start measurement of device's power/temp
    initial setup of nvml and vars
  */
  void setupSampling()
  {

    //reserve about 6 min of sampling space
    powerData.clear();
    powerData.reserve(750000);  
    tempData.clear();
    tempData.reserve(750000);  

    CUDA_ERROR( hipEventCreate(&gpuStart) );
    CUDA_ERROR( hipEventCreate(&gpuStop) );
    CUDA_ERROR( hipEventRecord(gpuStart) );
  }


  /*
  takeMeasurements
    loop that samples until kernel has finished
    return bool - true: good sample graph, false: bad samples
  */
  void runSampling() {
    unsigned int powerLevel, tempLevel;
    CUDA_ERROR( hipEventCreate(&kernelFinished) );
    CUDA_ERROR( hipEventRecord(kernelFinished) ); 

    while (hipEventQuery(kernelFinished) != hipSuccess) {
      powerLevel = getPowerUseage();
      tempLevel = getDeviceTemp();
      powerData.push_back(powerLevel);
      tempData.push_back(tempLevel);
    }
    CUDA_ERROR( hipEventRecord(gpuStop, 0) );
    CUDA_ERROR( hipEventSynchronize(gpuStop) );
  }


  /*
  get a good test reading
    keep running test until:
      good reading && temp was constant
      haven't looped too many times
    warn if test case is not long enough
    print each time test case is run
    if loop runs to maxItr, then print to file anyways
  */
  
  void getGoodSample() {
    unsigned int prevTemp = 0;
    unsigned int curTemp = 0;
    bool badSampleData = true;
    int curRun = 1;

    testClass->kernelSetup(deviceProp);

    while( badSampleData || std::abs((int)(curTemp - prevTemp)) >= 1) { 
      if (curRun > maxTestRuns ) {
        break;
      }
      printf("  beginning test run %d\n", curRun);
      //take temp measurement again here?
      setupSampling();
      testClass->runKernel();
      runSampling();

      prevTemp = curTemp;
      curTemp = getDeviceTemp();

      if (std::abs((int)(curTemp-prevTemp)) >= 1) {
        printf("    temp still changing\n");
      } else {
        badSampleData = !isDataValid();
      }
      curRun++;
    }

    if (maxTestRuns < curRun) {
      printf("maxTestRuns exceeded, sample data may be bad. Debug info:\n");
      printf("  curTemp - prevTemp = %d\n", (int)curTemp-prevTemp);
      printf("  badSampleData: %d\n", badSampleData);
    }
  }

  /*
  analyze test's sampling
    make sure graph is flat and level
    1%-3% tolerance in graph's height
      maybe allow small downward spikes 5ms in width
      very beg and end can ramp up/down though
  return true(good enough) / false(bad samples)
  */
  bool isDataValid() {
    if ((int)powerData.size() < 2*ignoreSampleCount) {
      printf("Only %d samples in previous run\n", (int)powerData.size());
      return false;
    }
    int startPt = ignoreSampleCount;
    int endPt = (int) powerData.size() - ignoreSampleCount;
    double avg = getPowerAvg(startPt, endPt);

    int badStreak = 0;
    double curSample;
    for (int i = startPt; i < endPt; i++) {
      curSample = (double) powerData[i];
      if (std::abs((curSample-avg)/avg) > 0.03) {

        if (badStreak++ > 250) {
          printf("    power samples not consistant enough. Made it %d%% through data\n", 100*i/endPt);
          return false;
        }

      } else {
        badStreak = 0;
      }
    }

    return true;
  }

  /*
  find the avg power value by random sampling
  */
  double getPowerAvg(int startPt, int endPt) {
    double total = 0;
    for (int i = startPt; i < endPt; i++) {
      total += (double)powerData[i];
    }
    return total / (endPt - startPt);
  }

  /*
  write data to file
  file is .txt but organized as .csv
    first row: "Power(W), Temperature(ºC), Time(ms), elapsed time(ms), number of samples"
    second row: "xxx, yyy, zzz, aaa, bb"
    third row->end: "xxxx, yyyy, zzzz"
  */
  void dataToFile() {
    float elapsedT;
    CUDA_ERROR( hipEventElapsedTime(&elapsedT, gpuStart, gpuStop));
    float timeBetweenSample = elapsedT / powerData.size();
    
    FILE *fp = fopen(outputName, "w+");
    if (fp == NULL) {
      printf("Attempt at opening '%s' failed. Error: ", outputName);
      perror("");
      printf("Terminating...");
      exit(0);
    }
    fprintf(fp, "Power(W), Temperature(ºC), Time(ms), elapsed time(ms), number of samples\n");
    fprintf(fp, "%.3lf, %d, %f, %3.1f, %d\n", powerData[0]/1000.0, tempData[0], timeBetweenSample, elapsedT, (int)powerData.size());
    
    for (int i = 1; i < powerData.size(); i++) {
      fprintf(fp, "%.3lf, %d, %f\n", powerData[i]/1000.0, tempData[0], timeBetweenSample*(i+1) );
    }
    fclose(fp);
  }

  /*
  find number of devices
  */
  void numOfDevices() {
    unsigned int deviceCount;
    nvmlResult = nvmlDeviceGetCount(&deviceCount);
    if (NVML_SUCCESS != nvmlResult)
    {
      printf("Failed to query device count: %s\n", nvmlErrorString(nvmlResult));
      exit(0);
    }
    printf("There are %d GPUs connected, ensure the correct one is selected\n", deviceCount);
  }

  /*
  get device temp
    return temp as unsigned int
    return 0 if there was a sampling error
  */
  unsigned int getDeviceTemp() {
    unsigned int tempLevel;
    nvmlResult = nvmlDeviceGetTemperature(nvmlDeviceID, NVML_TEMPERATURE_GPU, &tempLevel);
    if (nvmlResult != NVML_SUCCESS) {
      tempLevel = 0;
      printf("NVML temp measurement fail: %s\n", nvmlErrorString(nvmlResult));
    }
    return tempLevel;
  }

  unsigned int getPowerUseage() {
    unsigned int powerLevel;
    nvmlResult = nvmlDeviceGetPowerUsage(nvmlDeviceID, &powerLevel);
    if (nvmlResult != NVML_SUCCESS) {
      printf("NVML power measurement fail: %s\n", nvmlErrorString(nvmlResult));
    }
    return powerLevel;
  }

  void CUDA_ERROR(hipError_t e) {
    if (e != hipSuccess) {
      printf("cuda Error: \"%s\"\n", hipGetErrorString(e));
    }
  } 

}; //end TestRunner

template <typename T>
void runAddFPTest(int iterNum, int blockSize, const char* outputName1, 
              const char* outputName2) 
{
  printf("creating AdditionFP32_1 TestClass\n");
  AdditionFP_1<T> test1(blockSize, iterNum);
  printf("creating TestRunner obj\n");
  TestRunner<AdditionFP_1<T>> tester1(&test1, outputName1);
  printf("calling getGoodSample\n");
  tester1.getGoodSample();
  printf("calling dataToFile\n");
  tester1.dataToFile();

  printf("AdditionFP32_1 finished\n");

  printf("ceating AdditionFP32_2 TestClass\n");
  AdditionFP_2<T> test2(blockSize, iterNum);
  printf("creating TestRunner obj\n");
  TestRunner<AdditionFP_2<T>> tester2(&test2, outputName2);
  printf("calling getGoodSample\n");
  tester2.getGoodSample();
  printf("calling dataToFile\n");
  tester2.dataToFile();

  printf("AdditionFP32_2 finished\n");
}

int main() {
  printf("---- beginning FP32 Add Testing ----\n"); 
  runAddFPTest<float>(6000000, 256, "data/outputAddFP32_1.txt", "data/outputAddFP32_2.txt");
  printf("---- test end ----\n");

  printf("---- beginning FP64 Add Testing ----\n");
  runAddFPTest<double>(6000000, 256, "data/outputAddFP64_1.txt", "data/outputAddFP64_2.txt");
  printf("---- test end ----\n");

  printf("---- beginning Int32 Add Testing ---\n");
  runAddFPTest<int>(6000000, 256, "data/outputAddInt32_1.txt", "data/outputAddInt32_2.txt");
  printf("---- test end ----\n");
  return 0;
}
